#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Notice: this function could only be used to detect Cuda function whose return value type is not void. 
// we could use cudaGetLastError() to detect such cuda function (ex, self_define kernel function.)
hipError_t checkCuda(hipError_t result){
    if(result != hipSuccess){
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;

}

hipError_t checkLastCuda(){
    hipError_t result = hipGetLastError();    // return the error from above
    if(result != hipSuccess){
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }

    return result;
}